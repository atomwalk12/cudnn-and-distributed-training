#include "alexnet.h"
#include <cstdlib>

AlexNet::AlexNet(hipdnnHandle_t& handle, int batch_size, int num_classes) 
    : cudnn(handle), batch_size(batch_size), output_size(num_classes) {
    createNetwork();
}

void AlexNet::createNetwork() {
    // First convolution layer: 96 kernels of 11x11, stride 4
    layers.push_back(new ConvolutionLayer(cudnn, batch_size, 3, 96, 11, 4, 0));
    
    // Allocate memory for intermediate outputs
    // We'll need to calculate the output size for each layer
    // This is a placeholder - actual sizes need to be computed
    layer_outputs.push_back(nullptr);  // Will be set in forward pass
}

void AlexNet::forward(float *inp, float *out) {
    float* current_input = inp;
    
    for (size_t i = 0; i < layers.size(); i++) {
        float* current_output = (i == layers.size() - 1) ? out : layer_outputs[i];
        layers[i]->forward(current_input, current_output);
        current_input = current_output;
    }
}

float* AlexNet::createDummyGradient(float* output) {
    // Create a gradient of ones, similar to PyTorch's ones_like
    size_t output_dim = batch_size * output_size; // hardcoded for now, should match output size
    float* gradient;
    hipMallocManaged(&gradient, output_dim * sizeof(float));
    
    for (size_t i = 0; i < output_dim; i++) {
        gradient[i] = 1.0f;
    }
    return gradient;
}

void AlexNet::backwardInput(float* inp_grad, float* out_grad) {
    float* current_output_grad = out_grad;
    
    // Backward pass through layers in reverse order
    for (int i = layers.size() - 1; i >= 0; i--) {
        ConvolutionLayer* conv_layer = static_cast<ConvolutionLayer*>(layers[i]);
        float* current_input_grad = (i == 0) ? inp_grad : layer_outputs[i-1];
        conv_layer->backwardInput(current_input_grad, current_output_grad);
        current_output_grad = current_input_grad;
    }
}

void AlexNet::backwardParams(float* inp, float* out_grad) {
    float* current_input = inp;
    float* current_output_grad = out_grad;
    
    // Compute parameter gradients for each layer
    for (int i = layers.size() - 1; i >= 0; i--) {
        ConvolutionLayer* conv_layer = static_cast<ConvolutionLayer*>(layers[i]);
        conv_layer->backwardParams(current_input, current_output_grad);
        
        // Update input and gradient for next layer
        if (i > 0) {
            current_input = layer_outputs[i-1];
            current_output_grad = layer_outputs[i-1]; // This will be overwritten by backwardInput
        }
    }
}

AlexNet::~AlexNet() {
    // Delete all layers
    for (Layer* layer : layers) {
        delete layer;
    }
    
    // Free intermediate outputs
    for (float* output : layer_outputs) {
        if (output != nullptr) {
            hipFree(output);
        }
    }
}