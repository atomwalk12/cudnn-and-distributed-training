#include "utils.h"
#include "conv_layer.h"
#include <stdio.h>

ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t& cudnn_handle,
                                 int batch_size,
                                 int in_channels,
                                 int out_channels,
                                 int kernel_size,
                                 int stride,
                                 int padding)
    : Layer(cudnn_handle),
      batch_size(batch_size),
      in_channels(in_channels),
      out_channels(out_channels),
      kernel_size(kernel_size),
      stride(stride),
      padding(padding) {
    createDescriptors();
}

void ConvolutionLayer::createDescriptors() {
    // Add these as class members to track dimensions
    input_height = 224;  // Add to header
    input_width = 224;   // Add to header
    
    // Input descriptor
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnSetTensor4dDescriptor(
        input_descriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        batch_size,
        in_channels,
        input_height,
        input_width
    );

    // Filter descriptor
    hipdnnCreateFilterDescriptor(&filter_descriptor);
    hipdnnSetFilter4dDescriptor(
        filter_descriptor,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW,
        out_channels,
        in_channels,
        kernel_size,
        kernel_size
    );

    // Convolution descriptor
    hipdnnCreateConvolutionDescriptor(&conv_descriptor);
    hipdnnSetConvolution2dDescriptor(
        conv_descriptor,
        padding,
        padding,
        stride,
        stride,
        1,  // dilation_h
        1,  // dilation_w
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT
    );

    // Calculate output dimensions
    int out_n, out_c, out_h, out_w;
    hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w
    );

    // Store output dimensions as class members
    output_height = out_h;  // Add to header
    output_width = out_w;   // Add to header

    // Output descriptor
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnSetTensor4dDescriptor(
        output_descriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        out_n,
        out_c,
        out_h,
        out_w
    );

    // Allocate and initialize weights and biases
    size_t weight_size = out_channels * in_channels * kernel_size * kernel_size * sizeof(float);
    hipMallocManaged(&weights, weight_size);
    hipMallocManaged(&bias, out_channels * sizeof(float));
    hipMallocManaged(&weight_gradients, weight_size);

    // Initialize weights with random values
    for (size_t i = 0; i < weight_size/sizeof(float); i++) {
        weights[i] = (float)rand() / RAND_MAX;
    }

    // Initialize bias to zero
    for (int i = 0; i < out_channels; i++) {
        bias[i] = 0.0f;
    }

    // Debug print
    printf("Input dimensions: %dx%dx%dx%d\n", batch_size, in_channels, input_height, input_width);
    printf("Output dimensions: %dx%dx%dx%d\n", out_n, out_c, output_height, output_width);
    

    debugDescriptor("Input", input_descriptor);
    debugDescriptor("Output", output_descriptor);
    debugFilterDescriptor(filter_descriptor);

    // Debug print first few weights
    float debug_weights[10];
    hipMemcpy(debug_weights, weights, 10 * sizeof(float), hipMemcpyDeviceToHost);
    printf("First 10 weights: ");
    for(int i = 0; i < 10; i++) {
        printf("%.4f ", debug_weights[i]);
    }
    printf("\n");
    fflush(stdout);
}

void ConvolutionLayer::forward(float* input, float* output) {
    // Debug print first few input values
    float debug_input[10];
    hipMemcpy(debug_input, input, 10 * sizeof(float), hipMemcpyDeviceToHost);
    printf("First 10 inputs: ");
    for(int i = 0; i < 10; i++) {
        printf("%.4f ", debug_input[i]);
    }
    printf("\n");
    fflush(stdout);
    
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Implements: y = w ⊗ x
    // To simplify things, biases are ignored
    hipdnnStatus_t status = hipdnnConvolutionForward(
        cudnn,
        &alpha,
        input_descriptor,
        input,
        filter_descriptor,
        weights,
        conv_descriptor,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
        nullptr,  // workspace (we should add this as a class member)
        0,        // workspace size
        &beta,
        output_descriptor,
        output
    );
    if (status != HIPDNN_STATUS_SUCCESS) {
        printf("CUDNN forward failed: %s\n", hipdnnGetErrorString(status));
    }
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    // Debug print first few output values
    printf("First 10 outputs: ");
    float debug_output[10];
    hipMemcpy(debug_output, output, 10 * sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < 10; i++) {
        printf("%.4f ", debug_output[i]);
    }
    printf("\n");
    fflush(stdout);
}

void ConvolutionLayer::destroyDescriptors() {
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(conv_descriptor);
    hipFree(weights);
    hipFree(bias);
    hipFree(weight_gradients);
}

void ConvolutionLayer::backwardInput(float* input_gradient, float* output_gradient) {
    // First verify pointers
    if (input_gradient == nullptr || output_gradient == nullptr) {
        printf("Error: Null pointer passed to backwardInput\n");
        return;
    }

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Debug print input gradients
    float debug_outgrad[10];
    hipMemcpy(debug_outgrad, output_gradient, 10 * sizeof(float), hipMemcpyDeviceToHost);
    printf("First 10 output gradients: ");
    for(int i = 0; i < 10; i++) {
        printf("%.4f ", debug_outgrad[i]);
    }
    printf("\n");
    fflush(stdout);

    // Backward pass with different algorithm
    hipdnnStatus_t status = hipdnnConvolutionBackwardData(
        cudnn,
        &alpha,
        filter_descriptor,
        weights,
        output_descriptor,
        output_gradient,
        conv_descriptor,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1,  // Changed algorithm
        nullptr,
        0,
        &beta,
        input_descriptor,
        input_gradient
    );

    if (status != HIPDNN_STATUS_SUCCESS) {
        printf("Backward data failed: %s\n", hipdnnGetErrorString(status));
    }

    // Debug print output
    float debug_ingrad[10];
    hipMemcpy(debug_ingrad, input_gradient, 10 * sizeof(float), hipMemcpyDeviceToHost);
    printf("First 10 input gradients after backward: ");
    for(int i = 0; i < 10; i++) {
        printf("%.4f ", debug_ingrad[i]);
    }
    printf("\n");
    fflush(stdout);

    // Check for any CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error after backward: %s\n", hipGetErrorString(err));
    }
}

void ConvolutionLayer::backwardParams(float* input, float* output_gradient) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Implements: dL/dW = x ⊗ (dL/dy)
    hipdnnConvolutionBackwardFilter(
        cudnn,
        &alpha,
        input_descriptor,    // x descriptor
        input,               // x
        output_descriptor,   // dy descriptor
        output_gradient,     // dy
        conv_descriptor,     // convolution descriptor
        HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
        nullptr,            // workspace
        0,                  // workspace size
        &beta,
        filter_descriptor,  // dw descriptor
        weight_gradients    // dw
    );
}

void ConvolutionLayer::updateWeights(float learning_rate) {
    // Simple SGD update
    int weight_size = out_channels * in_channels * kernel_size * kernel_size;
    for(int i = 0; i < weight_size; i++) {
        weights[i] -= learning_rate * weight_gradients[i];
    }
}

ConvolutionLayer::~ConvolutionLayer() {
    destroyDescriptors();
}

