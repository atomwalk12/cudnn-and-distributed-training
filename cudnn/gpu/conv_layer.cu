#include "hip/hip_runtime.h"
#include "utils.h"
#include "conv_layer.h"
#include <stdio.h>
#include <hipblas.h>
#include <random>
#include <cmath>

ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t& cudnn_handle,
                                 int input_width,
                                 int input_height,
                                 int batch_size,
                                 int in_channels,
                                 int out_channels,
                                 int kernel_size,
                                 int stride,
                                 int padding)
    : Layer(cudnn_handle),
      batch_size(batch_size),
      in_channels(in_channels),
      out_channels(out_channels),
      kernel_size(kernel_size),
      stride(stride),
      padding(padding),
      input_height(input_height),
      input_width(input_width) {

    createDescriptors();
    calculateOutputDimensions();

    // Initialize cublas
    hipblasCreate(&cublas_handle);
}

void ConvolutionLayer::createDescriptors() {   
    // Input descriptor
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnSetTensor4dDescriptor(
        input_descriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        batch_size, in_channels, input_height, input_width
    );

    // Filter descriptor
    hipdnnCreateFilterDescriptor(&filter_descriptor);
    hipdnnSetFilter4dDescriptor(
        filter_descriptor,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW,
        out_channels, in_channels, kernel_size, kernel_size
    );

    // Convolution descriptor
    hipdnnCreateConvolutionDescriptor(&conv_descriptor);
    hipdnnSetConvolution2dDescriptor(
        conv_descriptor,
        padding,
        padding,
        stride,
        stride,
        1,  // dilation_h
        1,  // dilation_w
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT
    );

    // Calculate output dimensions
    int out_n, out_c, out_h, out_w;
    hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w
    );

    // Store output dimensions as class members
    output_height = out_h;  // Add to header
    output_width = out_w;   // Add to header

    // Output descriptor
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnSetTensor4dDescriptor(
        output_descriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        batch_size, out_channels, output_height, output_width
    );

    // Allocate and initialize weights and biases
    size_t weight_size = getWeightSize();
    hipMallocManaged(&weights, weight_size * sizeof(float));
    hipMallocManaged(&weight_gradients, weight_size * sizeof(float));

    // Initialize weights using Gaussian distribution (AlexNet paper specification)
    std::random_device rd;
    std::mt19937 gen(rd());
    float std = sqrt(2.0f / (in_channels * kernel_size * kernel_size));
    float std_alexnet = 0.01f;
    std::normal_distribution<float> distribution(0.0f, std);

    for (size_t i = 0; i < weight_size; i++) {
        weights[i] = distribution(gen);
    }

    debugDescriptor("Input", input_descriptor);
    debugDescriptor("Output", output_descriptor);
    debugFilterDescriptor(filter_descriptor);

    // Debug print first few weights
    debugTensorValues("weights", weights, 10);

    // Get workspace size needed for the selected algorithm
    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    workspace_size = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        algo,
        &workspace_size
    );

    // Allocate workspace memory
    workspace = nullptr;
    if (workspace_size > 0) {
        hipMalloc(&workspace, workspace_size);
    }
}

void ConvolutionLayer::forward(float* input, float* output) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform convolution using the pre-allocated workspace
    hipdnnStatus_t status = hipdnnConvolutionForward(
        cudnn,
        &alpha,
        input_descriptor,
        input,
        filter_descriptor,
        weights,
        conv_descriptor,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
        workspace,
        workspace_size,
        &beta,
        output_descriptor,
        output
    );
    if (status != HIPDNN_STATUS_SUCCESS) {
        printf("CUDNN forward failed: %s\n", hipdnnGetErrorString(status));
        exit(1);
    }
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Debug output values
    debugTensorValues("output", output, 10);
}

void ConvolutionLayer::destroyDescriptors() {
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(conv_descriptor);
    hipFree(weights);
    hipFree(bias);
    hipFree(weight_gradients);
}

void ConvolutionLayer::backwardInput(float* input_gradient, float* output_gradient) {
    // Debug output gradient values
    debugTensorValues("output gradient", output_gradient, 10);
    
    // Verify pointers
    if (input_gradient == nullptr || output_gradient == nullptr || weights == nullptr) {
        printf("Error: Null pointer in backwardInput\n");
        exit(1);
    }

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Get workspace size needed
    size_t workspace_size = 0;
    hipdnnStatus_t status = hipdnnGetConvolutionBackwardDataWorkspaceSize(
        cudnn,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1,
        &workspace_size
    );

    if (status != HIPDNN_STATUS_SUCCESS) {
        printf("Error getting workspace size: %s\n", hipdnnGetErrorString(status));
        exit(1);
    }

    // Allocate workspace
    void* workspace = nullptr;
    if (workspace_size > 0) {
        hipError_t err = hipMalloc(&workspace, workspace_size);
        if (err != hipSuccess) {
            printf("Workspace allocation failed: %s\n", hipGetErrorString(err));
            exit(1);
        }
    }

    // Backward pass with different algorithm
    // Implements: dL/dx = (dL/dy) @ w
    status = hipdnnConvolutionBackwardData(
        cudnn,
        &alpha,
        filter_descriptor,
        weights,
        output_descriptor,
        output_gradient,
        conv_descriptor,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1,
        workspace,        // Added workspace
        workspace_size,   // Added workspace size
        &beta,
        input_descriptor,
        input_gradient
    );

    if (status != HIPDNN_STATUS_SUCCESS) {
        printf("Backward data failed: %s\n", hipdnnGetErrorString(status));
        exit(1);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Free workspace
    if (workspace) {
        hipFree(workspace);
    }

    // Debug input gradient values
    debugTensorValues("input gradient", input_gradient, 10);
}

void ConvolutionLayer::backwardParams(float* input, float* output_gradient) {
    // Debug input and gradient values
    debugTensorValues("input", input, 10);
    debugTensorValues("output gradient", output_gradient, 10);
    
    // First verify pointers
    if (input == nullptr || output_gradient == nullptr || weight_gradients == nullptr) {
        printf("Error: Null pointer passed to backwardParams\n");
        return;
    }
    // Zero out gradients before computing new ones
    // TODO to delete
    // zeroGradients();

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Implements: dL/dW = x ⊗ (dL/dy)
    hipdnnStatus_t status = hipdnnConvolutionBackwardFilter(
        cudnn,
        &alpha,
        input_descriptor,
        input,
        output_descriptor,
        output_gradient,
        conv_descriptor,
        HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
        nullptr,
        0,
        &beta,
        filter_descriptor,
        weight_gradients
    );

    if (status != HIPDNN_STATUS_SUCCESS) {
        printf("Backward filter failed: %s\n", hipdnnGetErrorString(status));
        exit(1);
    }

    // Check for any CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error after backward filter: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Debug weight gradients
    debugTensorValues("weight gradients", weight_gradients, 10);
}

void ConvolutionLayer::zeroGradients() {
    hipMemset(weight_gradients, 0, getWeightSize() * sizeof(float));
}

ConvolutionLayer::~ConvolutionLayer() {
    destroyDescriptors();
    // Destroy cublas handle
    hipblasDestroy(cublas_handle);
    if (workspace) {
        hipFree(workspace);
    }
}

