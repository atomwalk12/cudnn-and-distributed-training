#include <stdio.h>
#include "alexnet.h"
#include <chrono>

// Benchmark parameters
const int NUM_ITERATIONS = 100;
const int WARMUP_ITERATIONS = 1;

void checkCUDNN(hipdnnStatus_t status) {
    if (status != HIPDNN_STATUS_SUCCESS) {
        printf("cuDNN Error: %s\n", hipdnnGetErrorString(status));
        exit(1);
    }
}

int main() {
    std::chrono::steady_clock::time_point begin, end;

    // Initialize CUDNN
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    // The data format is NCHW
    const int batch_size = 64;    // N
    const int channels = 3;       // C
    const int height = 224;       // H
    const int width = 224;        // W
    const int num_classes = 1000;

    // These vectors will be initialized with random values
    // ================================
    // =====      Input data      =====
    // ================================
    float *input_data, *output_data;
    hipMallocManaged(&input_data, batch_size*channels*height*width*sizeof(float));
    hipMallocManaged(&output_data, batch_size*num_classes*sizeof(float));

    for (int i = 0; i < batch_size * channels * height * width; i++) {
        input_data[i] = (float)rand() / RAND_MAX;
    }

    for (int i = 0; i < batch_size * num_classes; i++) {
        output_data[i] = 0.0f;
    }

    // Create and initialize the AlexNet model
    AlexNet model(cudnn, batch_size, num_classes);

    // Create dummy gradient for backward pass
    float* output_gradient = model.createDummyGradient(output_data);
    float* input_gradient;
    hipMallocManaged(&input_gradient, batch_size*channels*height*width*sizeof(float));
    hipDeviceSynchronize();

    // ================================
    // =====      Warmup run      =====
    // ================================
    for (int i = 0; i < WARMUP_ITERATIONS; i++) {
        model.forward(input_data, output_data);
        model.backwardInput(input_gradient, output_gradient);
        model.backwardParams(input_data, output_gradient);
    }
    hipDeviceSynchronize();

    // ================================
    // =====      Timing run      ===== 
    // ================================
    // =====     Forward pass     =====
    // ================================
    begin = std::chrono::steady_clock::now();
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        model.forward(input_data, output_data);
    }
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();

    double total_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    double average_milliseconds = (total_microseconds / 1000.0) / NUM_ITERATIONS;
    double total_time = average_milliseconds;
    printf("Average forward pass time: %f ms\n", average_milliseconds);

    // ================================
    // ===== Backward input pass ======
    // ================================
    begin = std::chrono::steady_clock::now();
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        model.backwardInput(input_gradient, output_gradient);
    }
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    
    total_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    average_milliseconds = (total_microseconds / 1000.0) / NUM_ITERATIONS;
    total_time += average_milliseconds;
    printf("Average backward input pass time: %f ms\n", average_milliseconds);
    
    // ================================
    // ===== Backward params pass =====
    // ================================
    begin = std::chrono::steady_clock::now();
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        model.backwardParams(input_data, output_gradient);
    }
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    
    total_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    average_milliseconds = (total_microseconds / 1000.0) / NUM_ITERATIONS;
    total_time += average_milliseconds;
    printf("Average backward params pass time: %f ms\n", average_milliseconds);
    printf("Total time: %f ms\n", total_time);

    // Additional cleanup
    hipFree(input_gradient);
    hipFree(output_gradient);

    // Cleanup
    hipFree(input_data);
    hipFree(output_data);
    hipdnnDestroy(cudnn);

    return 0;
}

