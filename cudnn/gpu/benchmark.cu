#include <stdio.h>
#include "alexnet.h"
#include <chrono>

void checkCUDNN(hipdnnStatus_t status) {
    if (status != HIPDNN_STATUS_SUCCESS) {
        printf("cuDNN Error: %s\n", hipdnnGetErrorString(status));
        exit(1);
    }
}

int main() {
    std::chrono::steady_clock::time_point begin, end;

    // Initialize CUDNN
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    // The data format is NCHW
    const int batch_size = 64;    // N
    const int channels = 3;       // C
    const int height = 224;       // H
    const int width = 224;        // W
    const int num_classes = 1000;

    // These vectors will be initialized with random values
    float *input_data, *output_data;
    hipMallocManaged(&input_data, batch_size*channels*height*width*sizeof(float));
    hipMallocManaged(&output_data, batch_size*num_classes*sizeof(float));

    for (int i = 0; i < batch_size * channels * height * width; i++) {
        input_data[i] = (float)rand() / RAND_MAX;
    }

    for (int i = 0; i < batch_size * num_classes; i++) {
        output_data[i] = 0.0f;
    }

    // Create and initialize the AlexNet model
    AlexNet model(cudnn, batch_size);

    // Cleanup
    hipFree(input_data);
    hipFree(output_data);

    return 0;
}

