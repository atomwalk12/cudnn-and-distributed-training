#include "utils.h"
#include <stdio.h>

void debugDescriptor(const char* name, hipdnnTensorDescriptor_t desc) {
    hipdnnDataType_t dtype;
    int n, c, h, w;
    int stride_n, stride_c, stride_h, stride_w;
    
    hipdnnGetTensor4dDescriptor(desc,
        &dtype,
        &n, &c, &h, &w,
        &stride_n, &stride_c, &stride_h, &stride_w);
        
    printf("%s descriptor:\n", name);
    printf("  Dimensions: %dx%dx%dx%d\n", n, c, h, w);
    printf("  Strides: %d,%d,%d,%d\n", stride_n, stride_c, stride_h, stride_w);
    fflush(stdout);
}

void debugFilterDescriptor(hipdnnFilterDescriptor_t desc) {
    hipdnnDataType_t dtype;
    hipdnnTensorFormat_t format;
    int k, c, h, w;
    
    hipdnnGetFilter4dDescriptor(desc,
        &dtype,
        &format,
        &k, &c, &h, &w);
        
    printf("Filter descriptor:\n");
    printf("  Dimensions: %dx%dx%dx%d\n", k, c, h, w);
    fflush(stdout);
} 